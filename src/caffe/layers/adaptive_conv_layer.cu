#include <vector>

#include "caffe/layers/adaptive_conv_layer.hpp"

namespace caffe {

template <typename Dtype>
void AdaptiveConvolutionLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* weights_up = this->blobs_[0]->gpu_data();
  const Dtype* weights_down = this->blobs_[1]->gpu_data();
  for (int i = 0; i < bottom.size(); ++i) {
    const Dtype* bottom_data = bottom[i]->gpu_data();
    Dtype* top_data = top[i]->mutable_gpu_data();
    for (int n = 0; n < this->num_; ++n) {
      this->forward_gpu_gemm(bottom_data + n * this->bottom_dim_, weights_up, weights_down,
          top_data + n * this->top_dim_);
      if (this->bias_term_) {
        const Dtype* bias = this->blobs_[2]->gpu_data();
        this->forward_gpu_bias(top_data + n * this->top_dim_, bias);
      }
    }
  }
}

template <typename Dtype>
void AdaptiveConvolutionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* weights_up = this->blobs_[0]->gpu_data();
  const Dtype* weights_down = this->blobs_[1]->gpu_data();
  Dtype* weight_diff_up = this->blobs_[0]->mutable_gpu_diff();
  Dtype* weight_diff_down = this->blobs_[1]->mutable_gpu_diff();
  Dtype* kernel_size_diff = this->blobs_[3]->mutable_gpu_diff();
  //caffe_gpu_set(this->kernel_shape_float_.count(),Dtype(0),kernel_size_diff);
  for (int i = 0; i < top.size(); ++i) {
    const Dtype* top_diff = top[i]->gpu_diff();
    // Bias gradient, if necessary.
    if (this->bias_term_ && this->param_propagate_down_[1]) {
      Dtype* bias_diff = this->blobs_[2]->mutable_gpu_diff();
      for (int n = 0; n < this->num_; ++n) {
        this->backward_gpu_bias(bias_diff, top_diff + n * this->top_dim_);
      }
    }
    if (this->param_propagate_down_[0] || propagate_down[i]) {
      const Dtype* bottom_data = bottom[i]->gpu_data();
      Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
      for (int n = 0; n < this->num_; ++n) {
        // gradient w.r.t. weight. Note that we will accumulate diffs.
        if (this->param_propagate_down_[0]) {
          this->weight_gpu_gemm(bottom_data + n * this->bottom_dim_,
              top_diff + n * this->top_dim_, weight_diff_up,weight_diff_down);
          this->backward_gpu_kernel_size(top_diff+n*this->top_dim_, bottom_data + n*this->bottom_dim_,
              weights_up,weights_down, kernel_size_diff);
        }
        // gradient w.r.t. bottom data, if necessary.
        if (propagate_down[i]) {
          this->backward_gpu_gemm(top_diff + n * this->top_dim_, weights_up, weights_down,
              bottom_diff + n * this->bottom_dim_);
        }
      }////
    }
  }
  this->update_kerneldiff_quene();
}

INSTANTIATE_LAYER_GPU_FUNCS(AdaptiveConvolutionLayer);

}  // namespace caffe
